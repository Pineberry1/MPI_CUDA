/*-----------------------------------------------
 * 请在此处填写你的个人信息
 * 学号:SA24011270 
 * 姓名:汤璇
 * 邮箱:txzzz@mail.ustc.edu.cn
 ------------------------------------------------*/

 
#include <hip/hip_runtime.h>
#include <chrono>
 #include <cstring>
 #include <fstream>
 #include <iostream>
 #include <string>
 
 #define AT(x, y, z) universe[(x) * N * N + (y) * N + z]
 
 using std::cin, std::cout, std::endl;
 using std::ifstream, std::ofstream;
 
 // 存活细胞数
 int population(int N, char *universe)
 {
     int result = 0;
     for (int i = 0; i < N * N * N; i++)
         result += universe[i];
     return result;
 }
 
 // 打印世界状态
 void print_universe(int N, char *universe)
 {
     // 仅在N较小(<= 32)时用于Debug
     if (N > 32)
         return;
     for (int x = 0; x < N; x++)
     {
         for (int y = 0; y < N; y++)
         {
             for (int z = 0; z < N; z++)
             {
                 if (AT(x, y, z))
                     cout << "O ";
                 else
                     cout << "* ";
             }
             cout << endl;
         }
         cout << endl;
     }
     cout << "population: " << population(N, universe) << endl;
 }
 
 __device__ int mod(int x, int n) {
     return (x + n) % n;
 }
 

__global__ void update(int N, char *universe, char* next) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    if (x >= N || y >= N || z >= N) return;

    extern __shared__ char shared_mem[];

    int shared_size = blockDim.x + 2;
    int local_x = threadIdx.x + 1;
    int local_y = threadIdx.y + 1;
    int local_z = threadIdx.z + 1;
    
    int global_idx = x * N * N + y * N + z;
    int shared_idx = local_x * shared_size * shared_size + local_y * shared_size + local_z;
    shared_mem[shared_idx] = universe[global_idx];

    // 边界面
    if (threadIdx.x == 0) {
        shared_mem[(local_x - 1) * shared_size * shared_size + local_y * shared_size + local_z] = universe[mod(x - 1, N) * N * N + y * N + z];
    }
    if (threadIdx.x == blockDim.x - 1 || x == N - 1) {
        shared_mem[(local_x + 1) * shared_size * shared_size + local_y * shared_size + local_z] = universe[mod(x + 1, N) * N * N + y * N + z];
    }
    if (threadIdx.y == 0) {
        shared_mem[local_x * shared_size * shared_size + (local_y - 1) * shared_size + local_z] = universe[x * N * N + mod(y - 1, N) * N + z];
    }
    if (threadIdx.y == blockDim.y - 1 || y == N - 1) {
        shared_mem[local_x * shared_size * shared_size + (local_y + 1) * shared_size + local_z] = universe[x * N * N + mod(y + 1, N) * N + z];
    }
    if (threadIdx.z == 0) {
        shared_mem[local_x * shared_size * shared_size + local_y * shared_size + (local_z - 1)] = universe[x * N * N + y * N + mod(z - 1, N)];
    }
    if (threadIdx.z == blockDim.z - 1 || z == N - 1) {
        shared_mem[local_x * shared_size * shared_size + local_y * shared_size + (local_z + 1)] = universe[x * N * N + y * N + mod(z + 1, N)];
    }

    // 边界边
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        shared_mem[(local_x - 1) * shared_size * shared_size + (local_y - 1) * shared_size + local_z] = universe[mod(x - 1, N) * N * N + mod(y - 1, N) * N + z];
    }
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1) {
        shared_mem[(local_x + 1) * shared_size * shared_size + (local_y + 1) * shared_size + local_z] = universe[mod(x + 1, N) * N * N + mod(y + 1, N) * N + z];
    }
    if (threadIdx.x == 0 && threadIdx.z == 0) {
        shared_mem[(local_x - 1) * shared_size * shared_size + local_y * shared_size + (local_z - 1)] = universe[mod(x - 1, N) * N * N + y * N + mod(z - 1, N)];
    }
    if (threadIdx.x == blockDim.x - 1 && threadIdx.z == blockDim.z - 1) {
        shared_mem[(local_x + 1) * shared_size * shared_size + local_y * shared_size + (local_z + 1)] = universe[mod(x + 1, N) * N * N + y * N + mod(z + 1, N)];
    }
    if (threadIdx.y == 0 && threadIdx.z == 0) {
        shared_mem[local_x * shared_size * shared_size + (local_y - 1) * shared_size + (local_z - 1)] = universe[x * N * N + mod(y - 1, N) * N + mod(z - 1, N)];
    }
    if (threadIdx.y == blockDim.y - 1 && threadIdx.z == blockDim.z - 1) {
        shared_mem[local_x * shared_size * shared_size + (local_y + 1) * shared_size + (local_z + 1)] = universe[x * N * N + mod(y + 1, N) * N + mod(z + 1, N)];
    }
    if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1) {
        shared_mem[(local_x - 1) * shared_size * shared_size + (local_y + 1) * shared_size + local_z] = universe[mod(x - 1, N) * N * N + mod(y + 1, N) * N + z];
    }
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0) {
        shared_mem[(local_x + 1) * shared_size * shared_size + (local_y - 1) * shared_size + local_z] = universe[mod(x + 1, N) * N * N + mod(y - 1, N) * N + z];
    }
    if (threadIdx.x == 0 && threadIdx.z == blockDim.z - 1) {
        shared_mem[(local_x - 1) * shared_size * shared_size + local_y * shared_size + (local_z + 1)] = universe[mod(x - 1, N) * N * N + y * N + mod(z + 1, N)];
    }
    if (threadIdx.x == blockDim.x - 1 && threadIdx.z == 0) {
        shared_mem[(local_x + 1) * shared_size * shared_size + local_y * shared_size + (local_z - 1)] = universe[mod(x + 1, N) * N * N + y * N + mod(z - 1, N)];
    }
    if (threadIdx.y == 0 && threadIdx.z == blockDim.z - 1) {
        shared_mem[local_x * shared_size * shared_size + (local_y - 1) * shared_size + (local_z + 1)] = universe[x * N * N + mod(y - 1, N) * N + mod(z + 1, N)];
    }
    if (threadIdx.y == blockDim.y - 1 && threadIdx.z == 0) {
        shared_mem[local_x * shared_size * shared_size + (local_y + 1) * shared_size + (local_z - 1)] = universe[x * N * N + mod(y + 1, N) * N + mod(z - 1, N)];
    }
    // 边界点（只在一个线程处理，省去很多分支判断）
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
        for (int corner = 0; corner < 8; ++corner) {
            int dx = (corner & 1) ? blockDim.x : -1;
            int dy = (corner & 2) ? blockDim.y : -1;
            int dz = (corner & 4) ? blockDim.z : -1;

            int neighbor_x = mod(x + dx, N);
            int neighbor_y = mod(y + dy, N);
            int neighbor_z = mod(z + dz, N);

            int global_neighbor_idx = neighbor_x * N * N + neighbor_y * N + neighbor_z;

            int local_neighbor_x = (dx == -1) ? 0 : (blockDim.x + 1);
            int local_neighbor_y = (dy == -1) ? 0 : (blockDim.y + 1);
            int local_neighbor_z = (dz == -1) ? 0 : (blockDim.z + 1);

            int shared_neighbor_idx = local_neighbor_x * shared_size * shared_size + 
                                      local_neighbor_y * shared_size + 
                                      local_neighbor_z;

            shared_mem[shared_neighbor_idx] = universe[global_neighbor_idx];
        }
    }

    __syncthreads();

    int alive = 0;
    for (int dx = -1; dx <= 1; ++dx) {
        for (int dy = -1; dy <= 1; ++dy) {
            for (int dz = -1; dz <= 1; ++dz) {
                if (dx == 0 && dy == 0 && dz == 0) continue;
                alive += shared_mem[(local_x + dx) * shared_size * shared_size + (local_y + dy) * shared_size + (local_z + dz)];
            }
        }
    }

    if (shared_mem[shared_idx] && (alive < 5 || alive > 7)) {
        next[global_idx] = 0;
    } else if (!shared_mem[shared_idx] && alive == 6) {
        next[global_idx] = 1;
    } else {
        next[global_idx] = shared_mem[shared_idx];
    }
}

 // 核心计算代码，将世界向前推进T个时刻
 const int blockSize = 8;
 __host__ void life3d_run(int N, char *universe, int T) {
     dim3 blockDim(blockSize, blockSize, blockSize);
     dim3 gridDim((N + blockSize - 1) / blockSize,
                  (N + blockSize - 1) / blockSize,
                  (N + blockSize - 1) / blockSize);
     size_t size = N * N * N * sizeof(char);
     char *universeInDevice, *next;
     hipMalloc(&universeInDevice, size);
     hipMalloc(&next, size);
     hipMemcpy(universeInDevice, universe, size, hipMemcpyHostToDevice);
 
     size_t shared_mem_size = (blockSize + 2) * (blockSize + 2) * (blockSize + 2) * sizeof(char);
 
     for (int t = 0; t < T; ++t) {
         update<<<gridDim, blockDim, shared_mem_size>>>(N, universeInDevice, next);
         hipDeviceSynchronize();
         std::swap(next, universeInDevice);
     }
 
     hipMemcpy(universe, universeInDevice, size, hipMemcpyDeviceToHost);
     hipFree(universeInDevice);
     hipFree(next);
 }
 
 // 读取输入文件
 void read_file(char *input_file, char *buffer)
 {
     ifstream file(input_file, std::ios::binary | std::ios::ate);
     if (!file.is_open())
     {
         cout << "Error: Could not open file " << input_file << std::endl;
         exit(1);
     }
     std::streamsize file_size = file.tellg();
     file.seekg(0, std::ios::beg);
     if (!file.read(buffer, file_size))
     {
         std::cerr << "Error: Could not read file " << input_file << std::endl;
         exit(1);
     }
     file.close();
 }
 
 // 写入输出文件
 void write_file(char *output_file, char *buffer, int N)
 {
     ofstream file(output_file, std::ios::binary | std::ios::trunc);
     if (!file)
     {
         cout << "Error: Could not open file " << output_file << std::endl;
         exit(1);
     }
     file.write(buffer, N * N * N);
     file.close();
 }
 
 int main(int argc, char **argv)
 {
     // cmd args
     if (argc < 5)
     {
         cout << "usage: ./life3d N T input output" << endl;
         return 1;
     }
     int N = std::stoi(argv[1]);
     int T = std::stoi(argv[2]);
     char *input_file = argv[3];
     char *output_file = argv[4];
 
     char *universe = (char *)malloc(N * N * N);
     read_file(input_file, universe);
 
     int start_pop = population(N, universe);
     auto start_time = std::chrono::high_resolution_clock::now();
     life3d_run(N, universe, T);
     auto end_time = std::chrono::high_resolution_clock::now();
     std::chrono::duration<double> duration = end_time - start_time;
     int final_pop = population(N, universe);
     write_file(output_file, universe, N);
 
     cout << "start population: " << start_pop << endl;
     cout << "final population: " << final_pop << endl;
     double time = duration.count();
     cout << "time: " << time << "s" << endl;
     cout << "cell per sec: " << T / time * N * N * N << endl;
 
     free(universe);
     return 0;
 }
 